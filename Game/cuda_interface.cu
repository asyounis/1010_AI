#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include "Global.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>    // For max()
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>




// __global__
__device__
void getGridForPiece(int piece, int &width, int &height, int pGrid[5][5])
{
    // Clear the array
    memset(pGrid, 0, sizeof(pGrid[0][0]) * 5 * 5);

    if (piece == 0)
    {
        width = 1;
        height = 3;
        pGrid[0][0] = 1;
        pGrid[1][0] = 1;
        pGrid[2][0] = 1;
    }
    else if (piece == 1)
    {
        width = 3;
        height = 1;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
        pGrid[0][2] = 1;
    }
    else if (piece == 2)
    {
        width = 3;
        height = 3;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
        pGrid[0][2] = 1;
        pGrid[1][2] = 1;
        pGrid[2][2] = 1;
    }
    else if (piece == 3)
    {
        width = 3;
        height = 3;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
        pGrid[0][2] = 1;
        pGrid[1][0] = 1;
        pGrid[2][0] = 1;
    }
    else if (piece == 4)
    {
        width = 3;
        height = 3;
        pGrid[0][0] = 1;
        pGrid[1][0] = 1;
        pGrid[2][0] = 1;
        pGrid[2][1] = 1;
        pGrid[2][2] = 1;
    }
    else if (piece == 5)
    {
        width = 3;
        height = 3;
        pGrid[0][2] = 1;
        pGrid[1][2] = 1;
        pGrid[2][0] = 1;
        pGrid[2][1] = 1;
        pGrid[2][2] = 1;
    }
    else if (piece == 6)
    {
        width = 1;
        height = 2;
        pGrid[0][0] = 1;
        pGrid[1][0] = 1;
    }
    else if (piece == 7)
    {
        width = 2;
        height = 1;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
    }
    else if (piece == 8)
    {
        width = 2;
        height = 2;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
        pGrid[1][0] = 1;
    }
    else if (piece == 9)
    {
        width = 2;
        height = 2;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
        pGrid[1][1] = 1;
    }
    else if (piece == 10)
    {
        width = 2;
        height = 2;
        pGrid[0][1] = 1;
        pGrid[1][1] = 1;
        pGrid[1][0] = 1;
    }
    else if (piece == 11)
    {
        width = 2;
        height = 2;
        pGrid[0][0] = 1;
        pGrid[1][1] = 1;
        pGrid[1][0] = 1;
    }
    else if (piece == 12)
    {
        width = 1;
        height = 5;
        pGrid[0][0] = 1;
        pGrid[1][0] = 1;
        pGrid[2][0] = 1;
        pGrid[3][0] = 1;
        pGrid[4][0] = 1;
    }
    else if (piece == 13)
    {
        width = 5;
        height = 1;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
        pGrid[0][2] = 1;
        pGrid[0][3] = 1;
        pGrid[0][4] = 1;
    }
    else if (piece == 14)
    {
        width = 1;
        height = 4;
        pGrid[0][0] = 1;
        pGrid[1][0] = 1;
        pGrid[2][0] = 1;
        pGrid[3][0] = 1;
    }
    else if (piece == 15)
    {
        width = 4;
        height = 1;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
        pGrid[0][2] = 1;
        pGrid[0][3] = 1;
    }
    else if (piece == 16)
    {
        width = 1;
        height = 1;
        pGrid[0][0] = 1;
    }
    else if (piece == 17)
    {
        width = 2;
        height = 2;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
        pGrid[1][0] = 1;
        pGrid[1][1] = 1;
    }
    else if (piece == 18)
    {
        width = 3;
        height = 3;
        pGrid[0][0] = 1;
        pGrid[0][1] = 1;
        pGrid[0][2] = 1;
        pGrid[1][0] = 1;
        pGrid[1][1] = 1;
        pGrid[1][2] = 1;
        pGrid[2][0] = 1;
        pGrid[2][1] = 1;
        pGrid[2][2] = 1;
    }
}

__device__
int processGrid(int grid[GAME_BOARD_GRID_SIZE][GAME_BOARD_GRID_SIZE], int x, int y, int pWidth, int pHeight)
{
    int linesCleared = 0;

    bool rowsToClear[GAME_BOARD_GRID_SIZE];
    bool columnsToClear[GAME_BOARD_GRID_SIZE];

    for (int i = y; i < y + pHeight; i++)
    {
        bool didFindHole = false;

        for (int j = 0; j < GAME_BOARD_GRID_SIZE; j++)
        {
            if (grid[i][j] == 0)
            {
                didFindHole = true;
                break;
            }
        }

        rowsToClear[i] = !didFindHole;
    }

    for (int i = x; i < x + pWidth; i++)
    {
        bool didFindHole = false;

        for (int j = 0; j < GAME_BOARD_GRID_SIZE; j++)
        {
            if (grid[j][i] == 0)
            {
                didFindHole = true;
                break;
            }
        }

        columnsToClear[i] = !didFindHole;
    }

    for (int i = y; i < y + pHeight; i++)
    {
        if (!rowsToClear[i])
        {
            continue;
        }

        linesCleared++;

        for (int j = 0; j < GAME_BOARD_GRID_SIZE; j++)
        {
            grid[i][j] = 0;
        }
    }

    for (int i = x; i < x + pWidth; i++)
    {
        if (!columnsToClear[i])
        {
            continue;
        }
        linesCleared++;

        for (int j = 0; j < GAME_BOARD_GRID_SIZE; j++)
        {
            grid[j][i] = 0;
        }
    }

    return linesCleared;
}

__device__
int isempty(int *stack, int &top) {

    if (top == -1)
        return 1;
    else
        return 0;
}

__device__
int peek(int *stack, int &top)  {
    return stack[top];
}

__device__
int pop(int *stack, int &top)  {
    int data;

    if (!isempty(stack, top)) {
        data = stack[top];
        top = top - 1;
        return data;
    } else {
        return -1;
    }
}

__device__
int isfull(int *stack, int &top)  {

    if (top == 100)
        return 1;
    else
        return 0;
}

__device__
void push(int *stack, int &top, int data) {

    if (!isfull(stack, top)) {
        top = top + 1;
        stack[top] = data;
    }
}


// Finds the maximum area under the histogram represented
// by histogram.  See below article for details.
// http://www.geeksforgeeks.org/largest-rectangle-under-histogram/
__device__
int maxHist(int row[GAME_BOARD_GRID_SIZE])
{
    /// Create an empty stack. The stack holds indexes of
    // hist[] array/ The bars stored in stack are always
    // in increasing order of their heights.
    // std::stack<int> result;
    int result[100];
    int top = 0;



    int top_val;     // Top of stack

    int max_area = 0; // Initialize max area in current
    // row (or histogram)

    int area = 0;    // Initialize area with current top

    // Run through all bars of given histogram (or row)
    int i = 0;
    while (i < GAME_BOARD_GRID_SIZE)
    {
        // If this bar is higher than the bar on top stack,
        // push it to stack
        if (isempty(result, top) || row[peek(result, top)] <= row[i])
            push(result, top, i++);

        else
        {
            // If this bar is lower than top of stack, then
            // calculate area of rectangle with stack top as
            // the smallest (or minimum height) bar. 'i' is
            // 'right index' for the top and element before
            // top in stack is 'left index'
            top_val = row[peek(result, top)];
            pop(result, top);
            area = top_val * i;

            if (!isempty(result, top))
                area = top_val * (i - peek(result, top) - 1 );
            max_area = max(area, max_area);
        }
    }

    // Now pop the remaining bars from stack and calculate area
    // with every popped bar as the smallest bar
    while (!isempty(result, top))
    {
        top_val = row[peek(result, top)];
        pop(result, top);
        area = top_val * i;
        if (!isempty(result, top))
            area = top_val * (i - peek(result, top) - 1 );

        max_area = max(area, max_area);
    }
    return max_area;
}

// Returns area of the largest rectangle with all 1s in A[][]
__device__
int maxRectangle(int A[GAME_BOARD_GRID_SIZE][GAME_BOARD_GRID_SIZE])
{
    // Calculate area for first row and initialize it as
    // result
    int result = maxHist(A[0]);

    // iterate over row to find maximum rectangular area
    // considering each row as histogram
    for (int i = 1; i < GAME_BOARD_GRID_SIZE; i++)
    {

        for (int j = 0; j < GAME_BOARD_GRID_SIZE; j++)

            // if A[i][j] is 1 then add A[i -1][j]
            if (A[i][j]) A[i][j] += A[i - 1][j];


        // Update result if area with current row (as last row)
        // of rectangle) is more
        result = max(result, maxHist(A[i]));
    }

    return result;
}

__device__
float evaluateMove(int grid[GAME_BOARD_GRID_SIZE][GAME_BOARD_GRID_SIZE], float heuristicCoeff[NUMBER_OF_HEURISTICS], int numberOfLinesCleared)
{
    int numberOfHoles = 0;
    int numberOfFreeLines = 0;
    int largestRectangleArea = 0;
    int totalNumberOfLineClusters = 0;

    bool freeRows[GAME_BOARD_GRID_SIZE];
    bool freeColumns[GAME_BOARD_GRID_SIZE];

    // Check Lines
    for (int x = 0; x < GAME_BOARD_GRID_SIZE; x++)
    {
        bool isFreeLine = true;
        for (int y = 0; y < GAME_BOARD_GRID_SIZE; y++)
        {
            if (grid[y][x] != 0)
            {
                isFreeLine = false;
                break;
            }
        }

        if (isFreeLine)
        {
            numberOfFreeLines++;
            freeColumns[x] = false;
        }
        else
        {
            freeColumns[x] = true;
        }
    }

    // Check Lines
    for (int y = 0; y < GAME_BOARD_GRID_SIZE; y++)
    {
        bool isFreeLine = true;
        for (int x = 0; x < GAME_BOARD_GRID_SIZE; x++)
        {
            if (grid[y][x] != 0)
            {
                isFreeLine = false;
                break;
            }
        }

        if (isFreeLine)
        {
            numberOfFreeLines++;
            freeRows[y] = true;
        }
        else
        {
            freeRows[y] = false;
        }
    }

    bool isOnCluster = false;
    for (int i = 0; i < GAME_BOARD_GRID_SIZE; i++)
    {
        if (freeRows[i] == false)
        {
            if (!isOnCluster)
            {
                isOnCluster = true;
                totalNumberOfLineClusters++;
            }
        }
        else
        {
            isOnCluster = false;
        }
    }

    isOnCluster = false;
    for (int i = 0; i < GAME_BOARD_GRID_SIZE; i++)
    {
        if (freeColumns[i] == false)
        {
            if (!isOnCluster)
            {
                isOnCluster = true;
                totalNumberOfLineClusters++;
            }
        }
        else
        {
            isOnCluster = false;
        }
    }

    // Number of Holes
    for (int x = 0; x < GAME_BOARD_GRID_SIZE; x++)
    {
        for (int y = 0; y < GAME_BOARD_GRID_SIZE; y++)
        {
            if (((x - 1) >= 0) && (grid[y][x - 1] == 0))
            {
                // Do nothing
            }
            else if (((y - 1) >= 0) && (grid[y - 1][x] == 0))
            {
                // Do nothing
            }
            else if (((y + 1) < GAME_BOARD_GRID_SIZE) && (grid[y + 1][x] == 0))
            {
                // Do nothing
            }
            else if (((x + 1) < GAME_BOARD_GRID_SIZE) && (grid[y][x + 1] == 0))
            {
                // Do nothing
            }
            else
            {
                // Non of the tests passed so its a hall
                numberOfHoles++;
            }

        }
    }

    // Get the largest rectangle
    largestRectangleArea = maxRectangle(grid);

    float moveScore = 0;
    moveScore =  (heuristicCoeff[0] * (float)numberOfLinesCleared);
    moveScore += (heuristicCoeff[1] * (float)numberOfFreeLines);
    moveScore += (heuristicCoeff[2] * (float)numberOfHoles);
    moveScore += (heuristicCoeff[3] * (float)largestRectangleArea);
    moveScore += (heuristicCoeff[4] * (float)totalNumberOfLineClusters);

    return moveScore;
}



__global__
void calculateMoves(int *grid,
                    int piece[NUMBER_OF_PIECES_PER_ROUND],
                    float heuristicCoeff[NUMBER_OF_HEURISTICS],
                    int *x,
                    int *y,
                    float *score,
                    int *linesCleared)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIndex >= (GAME_BOARD_GRID_SIZE * GAME_BOARD_GRID_SIZE))
    {
        for (int t = 0; t < NUMBER_OF_PIECES_PER_ROUND; t++)
        {
            x[threadIndex * NUMBER_OF_PIECES_PER_ROUND + t] = -1;
            y[threadIndex * NUMBER_OF_PIECES_PER_ROUND + t] = -1;
        }
        return;
    }


    return;

    score[threadIndex] = 0;
    linesCleared[threadIndex] = 0;


    int localGrid[GAME_BOARD_GRID_SIZE][GAME_BOARD_GRID_SIZE];
    for (int gX = 0; gX < GAME_BOARD_GRID_SIZE; gX++)
    {
        for (int gY = 0; gY < GAME_BOARD_GRID_SIZE; gY++)
        {
            localGrid[gY][gX] = grid[gY * GAME_BOARD_GRID_SIZE  + gX];
        }
    }


    int xPos[NUMBER_OF_PIECES_PER_ROUND];
    int yPos[NUMBER_OF_PIECES_PER_ROUND];

    int positionIndex = threadIndex;
    for (int i = 0; i < NUMBER_OF_PIECES_PER_ROUND; i++)
    {
        xPos[i] = positionIndex % GAME_BOARD_GRID_SIZE;
        positionIndex /= GAME_BOARD_GRID_SIZE;
        yPos[i] = positionIndex % GAME_BOARD_GRID_SIZE;
        positionIndex /= GAME_BOARD_GRID_SIZE;
    }

    // Get the piece data
    int pWidth = 0;
    int pHeight = 0;
    int pGrid[5][5];

    for (int i = 0; i < NUMBER_OF_PIECES_PER_ROUND; i++)
    {
        // Get the piece data
        getGridForPiece(piece[i], pWidth, pHeight, pGrid);

        bool isValidMove = true;

        // Check if it is a valid move
        for (int pX = 0; pX < pWidth; pX++)
        {
            for (int pY = 0; pY < pHeight; pY++)
            {
                if ((pGrid[pY][pX] == 1) && (localGrid[yPos[i] + pY][xPos[i] + pX] == 1))
                {
                    isValidMove = false;
                    break;
                }
            }

            if (!isValidMove)
            {
                break;
            }
        }


        if (!isValidMove)
        {
            for (int t = 0; t < NUMBER_OF_PIECES_PER_ROUND; t++)
            {
                x[threadIndex * NUMBER_OF_PIECES_PER_ROUND + t] = -1;
                y[threadIndex * NUMBER_OF_PIECES_PER_ROUND + t] = -1;
            }
            return;
        }

        x[threadIndex * NUMBER_OF_PIECES_PER_ROUND + i] = xPos[i];
        y[threadIndex * NUMBER_OF_PIECES_PER_ROUND + i] = yPos[i];

        // Place the piece since valid
        for (int pX = 0; pX < pWidth; pX++)
        {
            for (int pY = 0; pY < pHeight; pY++)
            {
                localGrid[yPos[i] + pY][xPos[i] + pX] += pGrid[pY][pX];
            }
        }

        linesCleared[threadIndex] += processGrid(localGrid, xPos[i] , yPos[i], pWidth, pHeight);
    }

    score[threadIndex] =  evaluateMove(localGrid, heuristicCoeff, linesCleared[threadIndex]);
}

extern "C"
int launchCuda(int grid[GAME_BOARD_GRID_SIZE][GAME_BOARD_GRID_SIZE],
               int piece[NUMBER_OF_PIECES_PER_ROUND],
               float heuristicCoeff[NUMBER_OF_HEURISTICS],
               int moveX[NUMBER_OF_PIECES_PER_ROUND],
               int moveY[NUMBER_OF_PIECES_PER_ROUND]
              )
{
    int numberOfMoves = 1;
    for (int i = 0; i < NUMBER_OF_PIECES_PER_ROUND; i++)
    {
        numberOfMoves *= GAME_BOARD_GRID_SIZE * GAME_BOARD_GRID_SIZE;
    }


    int *x;
    int *y;
    float *score;
    int *linesCleared;

    x = (int*)malloc(numberOfMoves * NUMBER_OF_PIECES_PER_ROUND * sizeof(int));
    y = (int*)malloc(numberOfMoves * NUMBER_OF_PIECES_PER_ROUND * sizeof(int));
    score = (float*)malloc(numberOfMoves * sizeof(float));
    linesCleared = (int*)malloc(numberOfMoves * sizeof(int));

    int *d_x;
    int *d_y;
    float *d_score;
    int *d_linesCleared;

    hipMalloc(&d_x, numberOfMoves * NUMBER_OF_PIECES_PER_ROUND * sizeof(int));
    hipMalloc(&d_y, numberOfMoves * NUMBER_OF_PIECES_PER_ROUND * sizeof(int));
    hipMalloc(&d_score, numberOfMoves * sizeof(float));
    hipMalloc(&d_linesCleared, numberOfMoves * sizeof(int));


    int *d_grid;
    int *d_piece;
    float *d_heuristicCoeff;

    hipMalloc(&d_grid, GAME_BOARD_GRID_SIZE * GAME_BOARD_GRID_SIZE * sizeof(int));
    hipMalloc(&d_piece, NUMBER_OF_PIECES_PER_ROUND * sizeof(int));
    hipMalloc(&d_heuristicCoeff, NUMBER_OF_HEURISTICS * sizeof(float));

    // hipMemcpy(d_grid, grid, GAME_BOARD_GRID_SIZE * GAME_BOARD_GRID_SIZE * sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(d_piece, piece, NUMBER_OF_PIECES_PER_ROUND * sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(d_heuristicCoeff, heuristicCoeff, NUMBER_OF_HEURISTICS * sizeof(float), hipMemcpyHostToDevice);


    // int threadsPerBlock = 128;

    // calculateMoves <<<(numberOfMoves / threadsPerBlock) + 1, threadsPerBlock>>>(d_grid,
    //                              d_piece,
    //                              d_heuristicCoeff,
    //                              d_x,
    //                              d_y,
    //                              d_score,
    //                              d_linesCleared);


    // hipMemcpy(x, d_x, numberOfMoves * NUMBER_OF_PIECES_PER_ROUND * sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(y, d_y, numberOfMoves * NUMBER_OF_PIECES_PER_ROUND * sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(score, d_score, numberOfMoves * sizeof(float), hipMemcpyDeviceToHost);
    // hipMemcpy(linesCleared, d_linesCleared, numberOfMoves * sizeof(int), hipMemcpyDeviceToHost);


    // int bestMove = -1;
    // float bestMoveScore = -10000000000;
    // for (int i = 0; i < numberOfMoves; i++)
    // {
    //     if (x[i * NUMBER_OF_PIECES_PER_ROUND] == -1)
    //     {
    //         continue;
    //     }

    //     if (score[i] > bestMoveScore)
    //     {
    //         bestMoveScore = score[i];
    //         bestMove = i;
    //     }
    // }

    // if (bestMove == -1)
    // {
    //     hipFree(d_grid);
    //     hipFree(d_piece);
    //     hipFree(d_heuristicCoeff);
    //     hipFree(d_x);
    //     hipFree(d_y);
    //     hipFree(d_score);
    //     hipFree(d_linesCleared);

    //     free(x);
    //     free(y);
    //     free(score);
    //     free(linesCleared);

    //     return 0;
    // }


    // for (int i = 0 ; i < NUMBER_OF_PIECES_PER_ROUND; i++)
    // {
    //     moveX[i] = x[bestMove * NUMBER_OF_PIECES_PER_ROUND + i];
    //     moveY[i] = y[bestMove * NUMBER_OF_PIECES_PER_ROUND + i];
    // }



    hipFree(d_grid);
    hipFree(d_piece);
    hipFree(d_heuristicCoeff);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_score);
    hipFree(d_linesCleared);

    free(x);
    free(y);
    free(score);
    free(linesCleared);

    return 1;
}